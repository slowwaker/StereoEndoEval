#include "hip/hip_runtime.h"
/*
Copyright (C) Chloe LeGendre

This program is free software; you can redistribute it and/or
modify it under the terms of the GNU General Public License
as published by the Free Software Foundation; either version 2
of the License, or (at your option) any later version.

This program is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
GNU General Public License for more details.

You should have received a copy of the GNU General Public License
along with this program; if not, write to the Free Software
Foundation, Inc., 51 Franklin Street, Fifth Floor, Boston, MA  02110-1301, USA.
*/

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>

#define TILE_WIDTH 32 

__global__ void SNCC_SummationFilter_Kernel_excl(float* device_costVol, float* device_costMap, unsigned char* device_disparityMap, int w, int h, int maxDisparity){
	
	int Row = blockIdx.y * TILE_WIDTH + threadIdx.y;
	int Col = blockIdx.x * TILE_WIDTH + threadIdx.x; 
	
	float A = 0;
	float B = 0;
	float C = 0;
	float D = 0;

	float corrScore = 0;
	float bestCorrScore = -RAND_MAX;
	int bestMatchSoFar = 0;
	
	if (Row > 5 && Col > 5 && Row < h - 7 && Col < w - 7){ //within summation filter bounds //was h-6, w-6 before
		for (int d = 0; d <= min(maxDisparity, Col - 6); d++){
			A = device_costVol[d*w*h + (Row - 6)*w + (Col - 6)]; // for exclusive scan. incl: [d*w*h + (Row - 7)*w + (Col - 7)];
			B = device_costVol[d*w*h + (Row - 6)*w + (Col + 7)]; // for exclusive scan. incl: [d*w*h + (Row - 7)*w + (Col + 6)];
			C = device_costVol[d*w*h + (Row + 7)*w + (Col - 6)]; // for exclusive scan. incl: [d*w*h + (Row + 6)*w + (Col - 7)];
			D = device_costVol[d*w*h + (Row + 7)*w + (Col + 7)]; // for exclusive scan. incl: [d*w*h + (Row + 6)*w + (Col + 6)];

			corrScore = (D - B - C + A)/169.0; //169 is for winAvg*winAvg

			if (bestCorrScore < corrScore){
					bestCorrScore = corrScore;
					bestMatchSoFar = d;
			}
		}

		device_costMap[Row*w + Col] = bestCorrScore;
		device_disparityMap[Row*w + Col] = unsigned char(bestMatchSoFar);
	}
}

__global__ void SNCC_DotProduct_KernelA(const float* __restrict__ device_patchMatrixLeft, const float* __restrict__ device_patchMatrixRight, float* __restrict__ device_costVol, int w, int h, int maxDisparity){

	int Row = blockIdx.y * TILE_WIDTH + threadIdx.y;
	int Col = blockIdx.x * TILE_WIDTH + threadIdx.x; 
	
	if (Row > 1 && Row < h - 2 && Col > 1 && Col < w - 2){ //within bounds for NCC window operation on image
	
		const int winsquared = 8;
		float leftTmp [winsquared];
		float NCC_val = 0.0;
		float rightTmp = 0.0;

		//load left vector
		for (int p = 0; p < winsquared; p++){
			leftTmp[p] = device_patchMatrixLeft[p*w*h + Row*w + Col];

		}

		//for all disparities
		for (int d = 0; d <= min(maxDisparity, Col - 2) ; d++){

			NCC_val = 0.0;
			for (int p = 0; p < winsquared; p++){
				rightTmp = device_patchMatrixRight[p*w*h + Row*w + Col - d];
				NCC_val += leftTmp[p]*rightTmp;
			}

			device_costVol[d*w*h + Row*w + Col] = NCC_val;
		}
	}
}

__global__ void SNCC_DotProduct_KernelB(const float* __restrict__ device_patchMatrixLeft, const float* __restrict__ device_patchMatrixRight, float* __restrict__ device_costVol, int w, int h, int maxDisparity){

	int Row = blockIdx.y * TILE_WIDTH + threadIdx.y;
	int Col = blockIdx.x * TILE_WIDTH + threadIdx.x; 
	
	if (Row > 1 && Row < h - 2 && Col > 1 && Col < w - 2){ //within bounds for NCC window operation on image
	
		const int winsquared = 16;
		float leftTmp [winsquared];
		float NCC_val = 0.0;
		float rightTmp = 0.0;

		//load left vector
		for (int p = 8; p < winsquared; p++){
			leftTmp[p] = device_patchMatrixLeft[p*w*h + Row*w + Col];

		}

		//for all disparities
		for (int d = 0; d <= min(maxDisparity, Col - 2) ; d++){ 

			NCC_val = 0.0;
			for (int p = 8; p < winsquared; p++){
				rightTmp = device_patchMatrixRight[p*w*h + Row*w + Col - d];
				NCC_val += leftTmp[p]*rightTmp;
			}

			device_costVol[d*w*h + Row*w + Col]+=NCC_val;
		}
	}
}

__global__ void SNCC_DotProduct_KernelC(const float* __restrict__ device_patchMatrixLeft, const float* __restrict__ device_patchMatrixRight, float* __restrict__ device_costVol, int w, int h, int maxDisparity){

	int Row = blockIdx.y * TILE_WIDTH + threadIdx.y;
	int Col = blockIdx.x * TILE_WIDTH + threadIdx.x; 
	
	if (Row > 1 && Row < h - 2 && Col > 1 && Col < w - 2){ //within bounds for NCC window operation on image
	
		const int winsquared = 25;
		float leftTmp [winsquared];
		float NCC_val = 0.0;
		float rightTmp = 0.0;

		//load left vector
		for (int p = 16; p < winsquared; p++){
			leftTmp[p] = device_patchMatrixLeft[p*w*h + Row*w + Col];

		}

		//for all disparities
		for (int d = 0; d <= min(maxDisparity, Col - 2) ; d++){ 

			NCC_val = 0.0;
			for (int p = 16; p < winsquared; p++){
				rightTmp = device_patchMatrixRight[p*w*h + Row*w + Col - d];
				NCC_val += leftTmp[p]*rightTmp;
			}

			device_costVol[d*w*h + Row*w + Col]+=NCC_val;
		}
	}
}

__global__ void SNCC_Mean_SD_Kernel(const unsigned char* __restrict__ device_left, const unsigned char* __restrict__ device_right, float* __restrict__ device_patchMatrixLeft, float* __restrict__ device_patchMatrixRight, int w, int h){

	int Row = blockIdx.y * TILE_WIDTH + threadIdx.y;
	int Col = blockIdx.x * TILE_WIDTH + threadIdx.x; 
	
	//create variable for shared memory
	__shared__ unsigned char tile_leftImage[TILE_WIDTH + 4][TILE_WIDTH + 4];
	__shared__ unsigned char tile_rightImage[TILE_WIDTH + 4][TILE_WIDTH + 4];

	//load left and right image data into shared memory
	if (Row < h && Col < w){
		tile_leftImage[threadIdx.y + 2][threadIdx.x + 2] = device_left[Row*w + Col];
		tile_rightImage[threadIdx.y + 2][threadIdx.x + 2] = device_right[Row*w + Col];
	}

	//load overlapping perimeter of block into shared memory (ghost elements)
	//LEFT pixels
	if (threadIdx.x == 0){
		if (Col > 1 && Row < h ){
			tile_leftImage[threadIdx.y + 2][threadIdx.x] = device_left[Row*w + Col - 2];
			tile_leftImage[threadIdx.y + 2][threadIdx.x + 1] = device_left[Row*w + Col - 1];
			tile_rightImage[threadIdx.y + 2][threadIdx.x] = device_right[Row*w + Col - 2];
			tile_rightImage[threadIdx.y + 2][threadIdx.x + 1] = device_right[Row*w + Col - 1];
		}
	}
	
	//RIGHT pixels
	if (threadIdx.x == TILE_WIDTH - 1){
		if (Col < (w - 2) && Row < h ){ 
			tile_leftImage[threadIdx.y + 2][threadIdx.x + 3] = device_left[Row*w + Col + 1];
			tile_leftImage[threadIdx.y + 2][threadIdx.x + 4] = device_left[Row*w + Col + 2];
			tile_rightImage[threadIdx.y + 2][threadIdx.x + 3] = device_right[Row*w + Col + 1];
			tile_rightImage[threadIdx.y + 2][threadIdx.x + 4] = device_right[Row*w + Col + 2];
		}
	}

	//TOP pixels
	if (threadIdx.y == 0){
		if (Row > 1 && Col < w ){
			tile_leftImage[threadIdx.y][threadIdx.x + 2] = device_left[(Row-2)*w + Col];
			tile_leftImage[threadIdx.y + 1][threadIdx.x + 2] = device_left[(Row-1)*w + Col];
			tile_rightImage[threadIdx.y][threadIdx.x + 2] = device_right[(Row-2)*w + Col];
			tile_rightImage[threadIdx.y + 1][threadIdx.x + 2] = device_right[(Row-1)*w + Col];
		}
	}

	//BOTTOM pixels
	if (threadIdx.y == TILE_WIDTH - 1){
		if (Row < h - 2 && Col < w ){
			tile_leftImage[threadIdx.y + 3][threadIdx.x + 2] = device_left[(Row+1)*w + Col];
			tile_leftImage[threadIdx.y + 4][threadIdx.x + 2] = device_left[(Row+2)*w + Col];
			tile_rightImage[threadIdx.y + 3][threadIdx.x + 2] = device_right[(Row+1)*w + Col];
			tile_rightImage[threadIdx.y + 4][threadIdx.x + 2] = device_right[(Row+2)*w + Col];
		}
	}
	
	//CORNERS
	//top left
	if (threadIdx.x == 0 && threadIdx.y == 0){
		if (Row > 1 && Col > 1){
			tile_leftImage[threadIdx.y][threadIdx.x] = device_left[(Row - 2)*w + Col - 2];
			tile_leftImage[threadIdx.y + 1][threadIdx.x] = device_left[(Row - 1)*w + Col - 2];
			tile_leftImage[threadIdx.y][threadIdx.x + 1] = device_left[(Row - 2)*w + Col - 1];
			tile_leftImage[threadIdx.y + 1][threadIdx.x + 1] = device_left[(Row - 1)*w + Col - 1];
			tile_rightImage[threadIdx.y][threadIdx.x] = device_right[(Row - 2)*w + Col - 2];
			tile_rightImage[threadIdx.y + 1][threadIdx.x] = device_right[(Row - 1)*w + Col - 2];
			tile_rightImage[threadIdx.y][threadIdx.x + 1] = device_right[(Row - 2)*w + Col - 1];
			tile_rightImage[threadIdx.y + 1][threadIdx.x + 1] = device_right[(Row - 1)*w + Col - 1];
		}
	}
	
	//top right
	if (threadIdx.x == TILE_WIDTH - 1 && threadIdx.y == 0){
		if (Row > 1 && Col < w - 2){
			tile_leftImage[threadIdx.y][threadIdx.x + 4] = device_left[(Row - 2)*w + Col + 2];
			tile_leftImage[threadIdx.y][threadIdx.x + 3] = device_left[(Row - 2)*w + Col + 1];
			tile_leftImage[threadIdx.y + 1][threadIdx.x + 3] = device_left[(Row - 1)*w + Col + 1];
			tile_leftImage[threadIdx.y + 1][threadIdx.x + 4] = device_left[(Row - 1)*w + Col + 2];
			tile_rightImage[threadIdx.y][threadIdx.x + 4] = device_right[(Row - 2)*w + Col + 2];
			tile_rightImage[threadIdx.y][threadIdx.x + 3] = device_right[(Row - 2)*w + Col + 1];
			tile_rightImage[threadIdx.y + 1][threadIdx.x + 3] = device_right[(Row - 1)*w + Col + 1];
			tile_rightImage[threadIdx.y + 1][threadIdx.x + 4] = device_right[(Row - 1)*w + Col + 2];
		}
	}
	
	//bottom left
	if (threadIdx.x == 0 && threadIdx.y == TILE_WIDTH - 1){
		if (Row < h - 2 && Col > 1){
			tile_leftImage[threadIdx.y + 4][threadIdx.x] = device_left[(Row + 2)*w + Col - 2];
			tile_leftImage[threadIdx.y + 3][threadIdx.x] = device_left[(Row + 1)*w + Col - 2];
			tile_leftImage[threadIdx.y + 4][threadIdx.x + 1] = device_left[(Row + 2)*w + Col - 1];
			tile_leftImage[threadIdx.y + 3][threadIdx.x + 1] = device_left[(Row + 1)*w + Col - 1];
			tile_rightImage[threadIdx.y + 4][threadIdx.x] = device_right[(Row + 2)*w + Col - 2];
			tile_rightImage[threadIdx.y + 3][threadIdx.x] = device_right[(Row + 1)*w + Col - 2];
			tile_rightImage[threadIdx.y + 4][threadIdx.x + 1] = device_right[(Row + 2)*w + Col - 1];
			tile_rightImage[threadIdx.y + 3][threadIdx.x + 1] = device_right[(Row + 1)*w + Col - 1];
			
		}
	}

	//bottom right
	if (threadIdx.x == TILE_WIDTH -1 && threadIdx.y == TILE_WIDTH - 1){
		if (Row < h - 2 && Col < w - 2){
			tile_leftImage[threadIdx.y + 4][threadIdx.x + 4] = device_left[(Row + 2)*w + Col + 2];
			tile_leftImage[threadIdx.y + 3][threadIdx.x + 4] = device_left[(Row + 1)*w + Col + 2];
			tile_leftImage[threadIdx.y + 3][threadIdx.x + 3] = device_left[(Row + 1)*w + Col + 1];
			tile_leftImage[threadIdx.y + 4][threadIdx.x + 3] = device_left[(Row + 2)*w + Col + 1];
			tile_rightImage[threadIdx.y + 4][threadIdx.x + 4] = device_right[(Row + 2)*w + Col + 2];
			tile_rightImage[threadIdx.y + 3][threadIdx.x + 4] = device_right[(Row + 1)*w + Col + 2];
			tile_rightImage[threadIdx.y + 3][threadIdx.x + 3] = device_right[(Row + 1)*w + Col + 1];
			tile_rightImage[threadIdx.y + 4][threadIdx.x + 3] = device_right[(Row + 2)*w + Col + 1];
		}
	}

	__syncthreads();

	//set boundaries to 0 in patchMatrixLeft and patchMatrixRight
	if ((Row == 0 || Row == 1) && Col < w ){ //top
		for (int d = 0; d < 25; d++){
			device_patchMatrixLeft[d*w*h + Row*w + Col] = 0;
			device_patchMatrixRight[d*w*h + Row*w + Col] = 0;
		}
	}
	else if ((Row == h - 1 || Row == h - 2) && Col < w){ //bottom
		for (int d = 0; d < 25; d++){
			device_patchMatrixLeft[d*w*h + Row*w + Col] = 0;
			device_patchMatrixRight[d*w*h + Row*w + Col] = 0;
		}
	}
	else if ((Col == 0 || Col == 1) && Row < h){//left
		for (int d = 0; d < 25; d++){
			device_patchMatrixLeft[d*w*h + Row*w + Col] = 0;
			device_patchMatrixRight[d*w*h + Row*w + Col] = 0;
		}	
	}
	else if ((Col == w - 1 || Col == w - 2) && Row < h){//right
		for (int d = 0; d < 25; d++){
			device_patchMatrixLeft[d*w*h + Row*w + Col] = 0;
			device_patchMatrixRight[d*w*h + Row*w + Col] = 0;
		}	
	}
	else {}

	//compute mean of each patch
	float leftMean = 0; 
	float rightMean = 0;
	float leftSD = 0; 
	float rightSD = 0;
	float winsquared = 25.0;
	int idx = 0;

	if ( Row < h - 2 && Row > 1 && Col < w - 2 && Col > 1){
		//iterate over window to get sum
		for (int p = threadIdx.y; p <= threadIdx.y + 4; p++ ){
			for (int q = threadIdx.x; q <= threadIdx.x + 4; q++){
				leftMean += float(tile_leftImage[p][q]);
				rightMean += float(tile_rightImage[p][q]);
			}
		}

		leftMean = leftMean/(winsquared);
		rightMean = rightMean/(winsquared);

		//iterate over window to get SD
		for (int p = threadIdx.y; p <= threadIdx.y + 4; p++ ){
			for (int q = threadIdx.x; q <= threadIdx.x + 4; q++){
				leftSD += pow((float(tile_leftImage[p][q]) - leftMean), 2);
				rightSD += pow((float(tile_rightImage[p][q]) - rightMean), 2);
			}
		}

		//iterate over window to subtract mean, divide by sqrt of sd
		for (int p = threadIdx.y; p <= threadIdx.y + 4; p++ ){
			for (int q = threadIdx.x; q <= threadIdx.x + 4; q++){
				if (leftSD == 0){
					device_patchMatrixLeft[idx*w*h + Row*w + Col] = 0;
				}
				else{
					device_patchMatrixLeft[idx*w*h + Row*w + Col] = ( float(tile_leftImage[p][q])  - leftMean)/sqrt(leftSD);
				}
				if (rightSD == 0){
					device_patchMatrixRight[idx*w*h + Row*w + Col] = 0;
				}
				else{
					device_patchMatrixRight[idx*w*h + Row*w + Col] = ( float(tile_rightImage[p][q]) - rightMean)/sqrt(rightSD);
				}
				idx++;
			}
		}
	}
}

__global__ void exclusive_scan_volume_kernelRow(const float* __restrict__ device_costVol, float* __restrict__ device_costVolIntegral, int w, int h){
	
	__shared__ float temp[2048];
	int tdx = threadIdx.x; 
	int offset = 1;
	int n = 2048;

	if (2*tdx+1 < w){
		temp[2*tdx] = device_costVol[ blockIdx.z*w*h + blockIdx.y*w + 2*tdx];
		temp[2*tdx+1] = device_costVol[ blockIdx.z*w*h + blockIdx.y*w + 2*tdx+1];
	}
	else if(2*tdx < w){
		temp[2*tdx] = device_costVol[ blockIdx.z*w*h + blockIdx.y*w + 2*tdx];
		temp[2*tdx+1] = 0;
	}
	else{
		temp[2*tdx] = 0;
		temp[2*tdx+1] = 0;
	}

	for(int d = n>>1; d > 0; d >>= 1){
		__syncthreads();
		if(tdx < d){
			int ai = offset*(2*tdx+1)-1;
			int bi = offset*(2*tdx+2)-1;
			temp[bi] += temp[ai];
		}
		offset <<= 1; //offset *= 2;
	}

	if(tdx == 0) temp[n - 1] = 0;
	for(int d = 1; d < n; d <<= 1){ //d *= 2
		offset >>= 1; __syncthreads();
		if(tdx < d){
			int ai = offset*(2*tdx+1)-1;
			int bi = offset*(2*tdx+2)-1;
			float t = temp[ai];
			temp[ai] = temp[bi];
			temp[bi] += t;
		}
	}

	__syncthreads();

	if (2*tdx+1 < w){
		device_costVolIntegral[ blockIdx.z*w*h + 2*tdx*h + blockIdx.y] = temp[2*tdx];
		device_costVolIntegral[ blockIdx.z*w*h + (2*tdx+1)*h + blockIdx.y] = temp[2*tdx+1];
	}
	else if (2*tdx < w){
		device_costVolIntegral[ blockIdx.z*w*h + 2*tdx*h + blockIdx.y] = temp[2*tdx];
	}
	else {} //do nothing
}

__global__ void exclusive_scan_volume_kernelCol(float* __restrict__ device_costVol, const float* __restrict__ device_costVolIntegral, int w, int h){
	
	__shared__ float temp[512];
	int tdx = threadIdx.x; 
	int offset = 1;
	int n = 512;

	if (2*tdx+1 < h){
		temp[2*tdx] = device_costVolIntegral[ blockIdx.z*w*h + blockIdx.x*h + 2*tdx ];
		temp[2*tdx+1] = device_costVolIntegral[ blockIdx.z*w*h + blockIdx.x*h + (2*tdx+1) ];

	}
	else if(2*tdx < h){
		temp[2*tdx] = device_costVolIntegral[ blockIdx.z*w*h + blockIdx.x*h + 2*tdx ];
		temp[2*tdx+1] = 0;
	}
	else{
		temp[2*tdx] = 0;
		temp[2*tdx+1] = 0;
	}

	for(int d = n>>1; d > 0; d >>= 1){
		__syncthreads();
		if(tdx < d){
			int ai = offset*(2*tdx+1)-1;
			int bi = offset*(2*tdx+2)-1;
			temp[bi] += temp[ai];
		}
		offset <<= 1; //offset *= 2;
	}

	if(tdx == 0) temp[n - 1] = 0;
	for(int d = 1; d < n; d <<= 1){ //d *= 2
		offset >>= 1; __syncthreads();
		if(tdx < d){
			int ai = offset*(2*tdx+1)-1;
			int bi = offset*(2*tdx+2)-1;
			float t = temp[ai];
			temp[ai] = temp[bi];
			temp[bi] += t;
		}
	}

	__syncthreads();

	if (2*tdx+1 < h){
		device_costVol[ blockIdx.z*w*h + 2*tdx*w + blockIdx.x] = temp[2*tdx];
		device_costVol[ blockIdx.z*w*h + (2*tdx+1)*w + blockIdx.x] = temp[2*tdx+1];
	}
	else if (2*tdx < h){
		device_costVol[ blockIdx.z*w*h + 2*tdx*w + blockIdx.x] = temp[2*tdx];
	}
	else {} //do nothing
}

int SNCC_Stereo_Matching_GPU(unsigned char *left, unsigned char *right, unsigned char *disparityImage, float *costMap, int w, int h, int win, int winAvg, int maxDisparity){
	
	std::cout << "in GPU SNCC function. Image size:" << w << " " << h << std::endl;
	
	if (win != 5 || winAvg != 13){
		std::cout << "Window size error. To use GPU, NCC window size must by 5x5 and summation window size must be 13x13."  << std::endl;
		hipDeviceReset();
		return 0;
	}

	//make sure we have a GPU
	int nDevices;
	hipError_t err = hipGetDeviceCount(&nDevices);
	if (err != hipSuccess) printf("%s\n", hipGetErrorString(err));

	//test GPU memory allocation for left and right stereo images
	int numBytesImage = w*h*sizeof(unsigned char);
	unsigned char *device_left = 0;
	unsigned char *device_right = 0;
	err = hipMalloc((void**)&device_left, numBytesImage);
	err = hipMalloc((void**)&device_right, numBytesImage);
	if (device_left == 0 || device_right == 0){
		printf("couldn't allocate memory for images \n");
		hipFree(device_left);
		hipFree(device_right);
	}
	if( err != hipSuccess)
    printf("hipMalloc error: %s\n", hipGetErrorString(err));

	//copy images to device
	err = hipMemcpy(device_left, left, numBytesImage, hipMemcpyHostToDevice);
	err = hipMemcpy(device_right, right, numBytesImage, hipMemcpyHostToDevice);
	if( err != hipSuccess)
    printf("hipMemcpy error: %s\n", hipGetErrorString(err));

	//test GPU memory allocation for patch matrices
	int numBytesPatch = w*h*(25)*sizeof(float); //25 is win*win
	float *device_patchMatrixLeft = 0; 
	float *device_patchMatrixRight = 0;
	err = hipMalloc((void**)&device_patchMatrixLeft, numBytesPatch);
	err = hipMalloc((void**)&device_patchMatrixRight, numBytesPatch);
	if (device_patchMatrixLeft == 0 || device_patchMatrixRight == 0){
		printf("couldn't allocate memory for patch matrices\n");
		hipFree(device_patchMatrixLeft);
		hipFree(device_patchMatrixRight);
		return 0;
	}
	if( err != hipSuccess)
    printf("hipMalloc error: %s\n", hipGetErrorString(err));

	//test GPU memory allocation for cost volume	
	int numBytesCostVol = w*h*(maxDisparity+1)*sizeof(float);
	float *device_costVol = 0;
	err = hipMalloc((void**)&device_costVol, numBytesCostVol);
	if (device_costVol == 0){
		printf("couldn't allocate memory for cost volume\n");
		hipFree(device_costVol);
		return 0;
	}
	if( err != hipSuccess)
    printf("hipMalloc error: %s\n", hipGetErrorString(err));

	//test GPU memory allocation for cost volume integral image
	float *device_costVolIntegral = 0;
	err = hipMalloc((void**)&device_costVolIntegral, numBytesCostVol);
	if (device_costVolIntegral == 0){
		printf("couldn't allocate memory for cost volume integral\n");
		hipFree(device_costVolIntegral);
		return 0;
	}
	if( err != hipSuccess)
    printf("hipMalloc error: %s\n", hipGetErrorString(err));
	
	//test GPU memory allocation for cost map
	int numBytesCostMap = w*h*sizeof(float);
	float *device_costMap = 0; 
	err = hipMalloc((void**)&device_costMap, numBytesCostMap);
	if (device_costMap == 0){
		printf("couldn't allocate memory for cost map \n");
		hipFree(device_costMap);
		return 0;
	}
	if( err != hipSuccess)
    printf("hipMalloc error: %s\n", hipGetErrorString(err));

	//test GPU memory allocation for disparity map
	int numBytesDisparityMap = w*h*sizeof(unsigned char);
	unsigned char *device_disparityMap = 0;
	err = hipMalloc((void**)&device_disparityMap, numBytesDisparityMap);
	if (device_disparityMap == 0){
		printf("coudln't allocate memory for disparity map\n");
		hipFree(device_disparityMap);
		return 0;
	}
	if( err != hipSuccess)
    printf("hipMalloc error: %s\n", hipGetErrorString(err));

	//setup execution configuration for first kernel: mean and SD kernel
	dim3 dimBlock(TILE_WIDTH,TILE_WIDTH);
	int gridx = int(ceil(float(w)/float(TILE_WIDTH)));
	int gridy = int(ceil(float(h)/float(TILE_WIDTH)));
	dim3 dimGrid(gridx, gridy);
	const unsigned int shared_mem_size = sizeof(unsigned char)*((4+TILE_WIDTH)*(4+TILE_WIDTH))*2; //*2 for left and right image

	//launch first kernel: mean and SD kernel
	SNCC_Mean_SD_Kernel<<<dimGrid, dimBlock, shared_mem_size>>>(device_left, device_right, device_patchMatrixLeft, device_patchMatrixRight, w, h);

	/*
	//uncomment this part for error checking kernel execution
	err=hipDeviceSynchronize();
	if( err != hipSuccess)
    printf("hipDeviceSynchronize error on Mean SD kernel: %s\n", hipGetErrorString(err));
	*/
	std::cout << "finished kernel to pre-compute vector mean and SD" << std::endl;
	
	//setup execution configuration for second kernel
	//preset all costVol bytes to 0
	err = hipMemset( device_costVol, 0,  numBytesCostVol);
	if( err != hipSuccess)
    printf("error on hipMemset 0: %s\n", hipGetErrorString(err));

	//launch second kernel: compute NCC volume over all possible disparity ranges
	SNCC_DotProduct_KernelA<<<dimGrid, dimBlock>>>(device_patchMatrixLeft, device_patchMatrixRight, device_costVol, w, h, maxDisparity);
	
	/*
	//uncomment this part for error checking kernel execution
	err=hipDeviceSynchronize();
	if( err != hipSuccess)
    printf("hipDeviceSynchronize error on Dot Product kernel A: %s\n", hipGetErrorString(err));
	*/
	std::cout << "finished kernel dot product A" << std::endl;
	
	SNCC_DotProduct_KernelB<<<dimGrid, dimBlock>>>(device_patchMatrixLeft, device_patchMatrixRight, device_costVol, w, h, maxDisparity);
	
	/*
	//uncomment this part for error checking kernel execution
	err=hipDeviceSynchronize();
	if( err != hipSuccess)
    printf("hipDeviceSynchronize error on Dot Product kernel B: %s\n", hipGetErrorString(err));
	*/
	std::cout << "finished kernel dot product B" << std::endl;
	
	SNCC_DotProduct_KernelC<<<dimGrid, dimBlock>>>(device_patchMatrixLeft, device_patchMatrixRight, device_costVol, w, h, maxDisparity);
	
	/*
	//uncomment this part for error checking kernel execution
	err=hipDeviceSynchronize();
	if( err != hipSuccess)
    printf("hipDeviceSynchronize error on Dot Product kernel C: %s\n", hipGetErrorString(err));
	*/
	std::cout << "finished kernel dot product C" << std::endl;

	//free patch matrix memory
	err = hipFree(device_patchMatrixLeft);
	err = hipFree(device_patchMatrixRight);
	if( err != hipSuccess)
    printf("hipFree error: %s\n", hipGetErrorString(err));

	//compute integral image on GPU, max width is 2048
	dim3 dimBlock2(1024,1,1);
	dim3 dimGrid2(1, h, maxDisparity);
	int shared_mem_size2 = 2048*sizeof(float);
	exclusive_scan_volume_kernelRow<<< dimGrid2, dimBlock2, shared_mem_size2>>>(device_costVol, device_costVolIntegral, w, h);
	
	/*
	//uncomment this part for error checking kernel execution
	err=hipDeviceSynchronize();
	if( err != hipSuccess)
    printf("hipDeviceSynchronize error on exclusive_scan_volume_kernelRow: %s\n", hipGetErrorString(err));
	*/
	std::cout << "finished exclusive scan kernel across rows" << std::endl;

	dim3 dimBlock3(256,1,1);
	dim3 dimGrid3(w, 1, maxDisparity);
	int shared_mem_size3 = 512*sizeof(float);
	exclusive_scan_volume_kernelCol<<< dimGrid3, dimBlock3, shared_mem_size3>>>(device_costVol, device_costVolIntegral, w, h);
	
	/*
	//uncomment this part for error checking kernel execution
	err=hipDeviceSynchronize();
	if( err != hipSuccess)
    printf("hipDeviceSynchronize error on exclusive_scan_volume_kernelCol: %s\n", hipGetErrorString(err));
	*/
	std::cout << "finished exclusive scan kernel across columns" << std::endl;
	
	//preset disparity map bytes to 0
	err = hipMemset( device_disparityMap, 0, numBytesDisparityMap);
	if( err != hipSuccess)
    printf("error on hipMemset 0: %s\n", hipGetErrorString(err));

	//launch third kernel: apply summation filter using integral image, based on exclusive scan
	SNCC_SummationFilter_Kernel_excl<<<dimGrid, dimBlock>>>(device_costVol, device_costMap, device_disparityMap, w, h, maxDisparity);
	
	/*
	//uncomment this part for error checking kernel execution
	err=hipDeviceSynchronize();
	if( err != hipSuccess)
    printf("hipDeviceSynchronize error on SNCC_SummationFilter_Kernel_excl: %s\n", hipGetErrorString(err));
	*/
	std::cout << "finished summation kernel" << std::endl;
	
	//copy device cost map and disparity map to host
	err = hipMemcpy(costMap, device_costMap, numBytesCostMap, hipMemcpyDeviceToHost);
	err = hipMemcpy(disparityImage, device_disparityMap, numBytesDisparityMap, hipMemcpyDeviceToHost); 
	if( err != hipSuccess)
    printf("hipMemcpy error: %s\n", hipGetErrorString(err));
	
	std::cout << "saving disparity map and cost map" << std::endl;

	hipFree(device_costVolIntegral);
	hipFree(device_costVol);
	hipFree(device_costMap);
	hipFree(device_disparityMap);
	hipFree(device_left);
	hipFree(device_right);
	
	hipDeviceReset();
	return 0;
}